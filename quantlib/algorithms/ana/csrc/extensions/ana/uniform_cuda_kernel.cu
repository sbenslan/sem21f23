#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>

// #include <stdio.h>  // for debug

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#define THREADS_PER_BLOCK 1024

#define ABS(x) ((x < 0.0f) ? -x : x)
#define CLAMP_0_1(x) ((x > 1.0f) ? 1.0f : ((x < 0.0f) ?  0.0f : x))


// GPU kernels (vanilla)

template <typename scalar_t>
__global__ void uniform_forward_cuda_kernel(
    scalar_t * const __restrict__ x_out,
    const scalar_t * __restrict__ x_in,
    const int64_t len_x,
    const scalar_t * __restrict__ t,
    const int64_t len_t,
    const scalar_t * __restrict__ q,
    const scalar_t * __restrict__ s_for,
    const scalar_t * __restrict__ training
)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix < len_x)
    {
        float sum = q[0];

        for (int it = 0; it < len_t; ++it)
        {
            // input position relative to the threshold
            float x_minus_t  = x_in[ix] - t[it];

            // expected value of the Heaviside function is the CDF of the uniform distribution
            float cdf;
            if (training && (*s_for != 0.0f))
            {
                float s_inv = 1.0f / (*s_for);
                cdf = CLAMP_0_1((0.5f * x_minus_t) * s_inv + 0.5f);
            }
            else
            {
                cdf = (float) (x_minus_t >= 0.0f); // use the Heaviside which maps zero to one
            }

            // dilate and accumulate expected step value
            float dq = q[it + 1] - q[it];
            sum += dq * cdf;
        }

        x_out[ix] = sum;
    }
    else  // I am out of bounds!
    {
        return;
    }
}


template <typename scalar_t>
__global__ void uniform_backward_cuda_kernel(
    scalar_t * const __restrict__ grad_out,
    const scalar_t * __restrict__ grad_in,
    const scalar_t * __restrict__ x_in,
    const int64_t len_x,
    const scalar_t * __restrict__ t,
    const int64_t len_t,
    const scalar_t * __restrict__ q,
    const scalar_t * __restrict__ s_back
)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix < len_x)
    {
        float sum = 0.0f;

        for (int it = 0; it < len_t; ++it)
        {
            // input position relative to the threshold
            float x_minus_t = x_in[ix] - t[it];

            // the derivative of the expected (i.e., regularised) step function is the PDF of the uniform distribution
            float pdf;
            if (*s_back != 0.0f)
            {
                float s_inv = 1.0f / (*s_back);
                float local_derivative = (float) (ABS(x_minus_t) <= (*s_back));
                pdf = 0.5f * s_inv * local_derivative;
            }
            else
            {
                pdf = 0.0f;  // no noise, no gradient!
            }

            // dilate and accumulate expected derivative
            float dq = q[it + 1] - q[it];
            sum += dq * pdf;
        }

        // compose gradients
        grad_out[ix] = sum * grad_in[ix];
    }
    else  // I am out of bounds!
    {
        return;
    }
}


// dispatchers

torch::Tensor uniform_forward_cuda_dispatch(
    torch::Tensor x_in,
    torch::Tensor t,
    torch::Tensor q,
    torch::Tensor s_for,
    torch::Tensor training
)
{
    auto x_out = torch::zeros_like(x_in);
    auto len_x = x_in.numel();

    const dim3 blocks((len_x + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

    AT_DISPATCH_FLOATING_TYPES(
        x_in.type(),
        "uniform_forward_cuda",
        ([&] {
            uniform_forward_cuda_kernel<scalar_t><<<blocks, THREADS_PER_BLOCK>>>(
                x_out.data_ptr<scalar_t>(),
                x_in.data_ptr<scalar_t>(),
                len_x,
                t.data_ptr<scalar_t>(),
                t.numel(),
                q.data_ptr<scalar_t>(),
                s_for.data_ptr<scalar_t>(),
                training.data_ptr<scalar_t>()
            );
        })
    );

    return x_out;
}


torch::Tensor uniform_backward_cuda_dispatch(
    torch::Tensor grad_in,
    torch::Tensor x_in,
    torch::Tensor t,
    torch::Tensor q,
    torch::Tensor s_back
)
{
    auto grad_out = torch::zeros_like(x_in);
    auto len_x = x_in.numel();

    const dim3 blocks((len_x + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

    AT_DISPATCH_FLOATING_TYPES(
        x_in.type(),
        "uniform_backward_cuda",
        ([&] {
            uniform_backward_cuda_kernel<scalar_t><<<blocks, THREADS_PER_BLOCK>>>(
                grad_out.data_ptr<scalar_t>(),
                grad_in.data_ptr<scalar_t>(),
                x_in.data_ptr<scalar_t>(),
                len_x,
                t.data_ptr<scalar_t>(),
                t.numel(),
                q.data_ptr<scalar_t>(),
                s_back.data_ptr<scalar_t>()
            );
        })
    );

    return grad_out;
}
